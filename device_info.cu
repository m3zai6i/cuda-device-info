/*

Compile: nvcc <filename>.cu
Execute: ./a.out

*/

#include <stdio.h>
#include <stdlib.h>

int main()
{
	int nrOfDevices;
	hipGetDeviceCount(&nrOfDevices);
	for (int device = 0; device < nrOfDevices; device++)
	{
		hipDeviceProp_t prp;
		hipGetDeviceProperties(&prp, device);
		printf("Device Number is: %d\n", device);
		printf("Device name is: %s\n", prp.name);
		printf("The memory Bus Width is %d bits\n", prp.memoryBusWidth);
		printf("Warp size: %d\n", prp.warpSize);
		printf("Maximum memory pitch: %ld\n", prp.memPitch);
		printf("Memory Clock Rate is %d KHz \n", prp.memoryClockRate);
		printf("Maximum threads per block is %d	\n", prp.maxThreadsPerBlock);
		printf("Peak Memory Bandwidth is %f GB/s \n", 2.0 *prp.memoryClockRate *(prp.memoryBusWidth / 8) / 1.0e6);
		printf("Number of multiprocessors is %d	\n", prp.multiProcessorCount);

	}

	return 0;
}
